
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define INIT_TIME(prev, init) \
    gettimeofday(&prev, NULL); \
    gettimeofday(&init, NULL);

// remove overhead created by call to gettimeofday
#define GET_TIME(prev, init, final, res) \
    gettimeofday(&final, NULL); \
    res = (final.tv_sec-init.tv_sec+(final.tv_usec-init.tv_usec)/1.e6) - \
          (init.tv_sec-prev.tv_sec+(init.tv_usec-prev.tv_usec)/1.e6);

#define SYNC \
    { \
        hipError_t err = hipDeviceSynchronize(); \
        if (err != hipSuccess) { \
            printf("Kernel launch failed: %s\n", hipGetErrorString(err)); \
        } \
    }

void _printMatrix(float *A, int m, int n, int p) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < p; k++){
                printf("%.2f ", A[i * n * p + j * p + k]);
            }
            printf("\n");
        }
        printf("\n");
    }
}

void _initMatrix(float *A, int m, int n, int p) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < p; k++) {
                A[i * n * p + j * p + k] = k + 1.0;
            }
        }
    }
}

__global__ void kernel1(float *A, int n, int p) {
    extern __shared__ float s[];
    int b_idx = blockIdx.x;
    int t_x = threadIdx.x;
    int t_y = threadIdx.y;

    s[t_x * p + t_y] = A[b_idx * n * p + t_x * p + t_y];

    // STEP 1
    float l_min = s[t_x * p + t_y];
    for (int offset = p / 2; offset > 0; offset /= 2) {
        l_min = min(l_min, __shfl_down_sync(0xFFFFFFFF, l_min, offset, p));
    }

    // TODO: try to use __shfl_sync instead of going through shared memory
    // l_min = __shfl_sync(0xFFFFFFFF, l_min, 0);
    l_min = s[t_x * p];

    s[t_x * p + t_y] += l_min;

    // sync because we need to make sure all threads have updated their values
    // before entering the next step
    __syncthreads();

    // STEP 2
    if (t_x == 0) {
        A[b_idx * n * p + t_x * p + t_y] = s[t_x * p + t_y] + s[(t_x + 1) * p + t_y];
    } else if (t_x == n - 1) {
        A[b_idx * n * p + t_x * p + t_y] = s[t_x * p + t_y] + s[(t_x - 1) * p + t_y];
    } else {
        A[b_idx * n * p + t_x * p + t_y] = s[t_x * p + t_y] + s[(t_x - 1) * p + t_y] + s[(t_x + 1) * p + t_y];
    }
}

__global__ void kernel2(float *A, int m, int n, int p) {
    int m_idx = blockIdx.x;
    int t_idx = threadIdx.x * p + threadIdx.y;

    // STEP 3
    float l_val = A[m_idx * n * p + t_idx];
    if (m_idx == 0) {
        l_val += A[(m_idx + 1) * n * p + t_idx];
    } else if (m_idx == m - 1) {
        l_val += A[(m_idx - 1) * n * p + t_idx];
    } else {
        l_val += A[(m_idx - 1) * n * p + t_idx] + A[(m_idx + 1) * n * p + t_idx];
    }
    A[m_idx * n * p + t_idx] = l_val;
}

int main(int argc, char *argv[]) {
    struct timeval t_prev, t_init, t_final;
    double kernel_t;

    if (argc < 4) {
        printf("Usage: %s M N P\n", argv[0]);
        return 1;
    }

    int M = atoi(argv[1]);
    int N = atoi(argv[2]);
    int P = atoi(argv[3]);

    unsigned int numBytes = M * N * P * sizeof(float);
    float *X = (float *)malloc(numBytes);

    if (!X || !numBytes) {
        printf("Memory allocation failed\n");
        return 1;
    }

    _initMatrix(X, M, N, P);

    printf("INPUT:\n");
    _printMatrix(X, M, N, P);

    float *dX;
    hipMalloc(&dX, numBytes);
    hipMemcpy(dX, X, numBytes, hipMemcpyHostToDevice);

    dim3 dimGrid(M);
    dim3 dimBlock(N, P);
    
    INIT_TIME(t_prev, t_init);
    kernel1<<<dimGrid, dimBlock, N * P * sizeof(float)>>>(dX, N, P);
    kernel2<<<dimGrid, dimBlock>>>(dX, M, N, P);
    SYNC;
    GET_TIME(t_prev, t_init, t_final, kernel_t);

    hipMemcpy(X, dX, numBytes, hipMemcpyDeviceToHost);

    printf("OUTPUT:\n");
    _printMatrix(X, M, N, P);

    hipFree(dX);
    free(X);

    FILE *fp = fopen((argc > 4) ? argv[4] : "out.csv", "a");
    fprintf(fp, "%d,%d,%d,%ld,%f,%d\n", M, N, P, N * P * sizeof(float), kernel_t, N * P);
    fclose(fp);

    return 0;
}
