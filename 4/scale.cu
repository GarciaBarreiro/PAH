
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <string.h>
#include <sys/time.h>

#ifndef DEBUG
    #define DEBUG 0
#else
    #define DEBUG 1
#endif

#define INIT_TIME(prev, init) \
    gettimeofday(&prev, NULL); \
    gettimeofday(&init, NULL);

// remove overhead created by call to gettimeofday
#define GET_TIME(prev, init, final, res) \
    gettimeofday(&final, NULL); \
    res = (final.tv_sec-init.tv_sec+(final.tv_usec-init.tv_usec)/1.e6) - \
          (init.tv_sec-prev.tv_sec+(init.tv_usec-prev.tv_usec)/1.e6);

void _printMat(float *A, int m, int n) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%.3f ", A[i * n + j]);
        }
        printf("\n");
    }
}

void _printVec(float *v, int n) {
    printf("[");
    for (int i = 0; i < n; i++) {
        printf("%.2f ", v[i]);
    }
    printf("\b]\n\n");
}

__global__ void factorMat(float *A, float *B, int m, int n, float factor) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < m * n) {
        B[i] = A[i] * factor;
    }
}

int main(int argc, char **argv) {
    struct timeval t_prev, t_init, t_final;
    double htran_t, kernel_t;  // host to device, device to host, and kernel time

    if (argc < 5) {
        printf("Usage: %s <m> <n> <rep> <factors>\n", argv[0]);
        return 1;
    }

    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int rep = atoi(argv[3]);
    char *f = argv[4]; // temp var
    float *factor = (float *)malloc(rep * sizeof(float));
    int thr_block = (argc > 5) ? atoi(argv[5]) : 32;      // threads per block

    // checks everything's OK, splits and saves to factor
    if (!(f[0] == '[' && f[strlen(f) - 1] == ']')) {
        printf("Factors are inputted like [0.1,0.2,0.6]\n");
        return 1;
    }

    // converts factor string to an array of floats
    f[strlen(f) - 1] = '\0';
    if (rep == 1) {
        factor[0] = atof(++f);
    } else {
        char *fact = strtok(++f, ",");
        for (int i = 0; i < rep; i++) {
            if (!fact) {
                printf("Not enough factors\n");
                return 1;
            }
            factor[i] = atof(fact);
            fact = strtok(NULL, ",");
        }
    }

    if (DEBUG) { _printVec(factor, rep); }

    unsigned int numBytes = m * n * sizeof(float);

    // init host
    float *cA = (float *)malloc(numBytes);  // in
    for (int i = 0; i < m * n; i++) { cA[i] = i; }
    float *cB = (float *)malloc(rep * numBytes);  // out
    int offset = m * n;

    // init device
    float *A;
    hipMalloc(&A, numBytes);
    float *B;
    hipMalloc(&B, rep * numBytes);

    INIT_TIME(t_prev, t_init);

    hipMemcpy(A, cA, numBytes, hipMemcpyHostToDevice);

    GET_TIME(t_prev, t_init, t_final, htran_t);

    dim3 dimBlock(thr_block);
    dim3 dimGrid((n * m + dimBlock.x - 1) / dimBlock.x);

    int n_streams = (rep < 8) ? rep : 8;    // default CUDA_DEVICE_MAX_CONNECTIONS value
    hipStream_t *streams = (hipStream_t *)malloc(n_streams * sizeof(hipStream_t));

    for (int i = 0; i < n_streams; i++) { hipStreamCreate(&streams[i]); }

    if (DEBUG) {
        printf("A:\n");
        _printMat(cA, m, n);
        printf("\n");
    }

    INIT_TIME(t_prev, t_init);

    for (int i = 0; i < rep; i++) {
        factorMat<<<dimGrid, dimBlock, 0, streams[i % n_streams]>>>(A, &B[i * offset], m, n, factor[i]);
        hipMemcpyAsync(&cB[i * offset], &B[i * offset], numBytes, hipMemcpyDeviceToHost, streams[i % n_streams]);
    }

    for (int i = 0; i < n_streams; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    GET_TIME(t_prev, t_init, t_final, kernel_t);

    for (int i = 0; i < rep; i++) {
        printf("%d >>>>>>>>>>>>>>>>>>> %.2f\n", i, factor[i]);
        _printMat(&cB[i * offset], m, n);
        printf("\n");
    }

    hipFree(A);

    free(factor);
    free(cA);
    free(cB);
    free(streams);

    // write results
    FILE *fp = fopen((argc > 6) ? argv[6] : "out.csv", "a");
    if (!fp) { printf("Error opening file\n"); }
    else {
        fprintf(fp, "%d,%d,%d,%d,%f,%f,%f\n", m, n, thr_block, rep, htran_t, kernel_t, htran_t + kernel_t);
        fclose(fp);
    }

    if (DEBUG) {
        printf("Host transfer time: %f\n", htran_t);
        printf("Kernel time: %f\n", kernel_t);
        printf("Total time: %f\n", htran_t + kernel_t);
    }

    return 0;
}
